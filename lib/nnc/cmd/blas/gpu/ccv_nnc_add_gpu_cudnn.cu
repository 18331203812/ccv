extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_add_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	float p = cmd.info.blas.a[0];
	float q = cmd.info.blas.a[1];
	static const float zero = 0;
	if (inputs[1] == 0)
	{
		const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
		const ccv_nnc_cudnn_tensor_view_descriptor_t c = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
		CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &p, a.descriptor, a.data.u8,  &zero, c.descriptor, c.data.u8));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(c);
		return CCV_NNC_EXEC_SUCCESS;
	}
	ccv_nnc_tensor_view_t atv = ccv_nnc_get_tensor_view(inputs[0]);
	ccv_nnc_tensor_view_t btv = ccv_nnc_get_tensor_view(inputs[1]);
	ccv_nnc_tensor_view_t* tvs[] = {
		&atv, &btv
	};
	ccv_nnc_tensor_view_alignment(tvs, 2);
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(&atv, adim);
	int bdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(&btv, bdim);
	// If the input a doesn't match the output. We can do two things:
	// 1. If b matches, we switch;
	// 2. Otherwise, we change a's dimension and stride.
	hipdnnOpTensorDescriptor_t add = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(add, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	ccv_nnc_cudnn_tensor_view_descriptor_t a;
	if (!ccv_nnc_tensor_view_check_dim((const ccv_nnc_tensor_view_t*)outputs[0], adim))
	{
		if (ccv_nnc_tensor_view_check_dim((const ccv_nnc_tensor_view_t*)outputs[0], bdim))
		{
			ccv_nnc_tensor_view_t t;
			CCV_SWAP(atv, btv, t);
			float v;
			CCV_SWAP(p, q, v);
			a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &atv);
		} else {
			const ccv_nnc_cudnn_tensor_view_descriptor_t old_a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &atv);
			void* const workspace = ccv_nnc_stream_context_get_workspace(stream_context, ccv_nnc_tensor_data_size(outputs[0]->info), CCV_TENSOR_GPU_MEMORY);
			ccv_nnc_tensor_t tensor = ccv_nnc_tensor(workspace, outputs[0]->info, 0);
			a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&tensor);
			static const float one = 1;
			CUDNN_ENFORCE(hipdnnOpTensor(cudnn, add, &zero, a.descriptor, a.data.u8, &one, old_a.descriptor, old_a.data.u8, &zero, a.descriptor, a.data.u8));
			ccv_nnc_cudnn_deinit_tensor_view_descriptor(old_a);
		}
	} else
		a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &atv);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &btv);
	const ccv_nnc_cudnn_tensor_view_descriptor_t c = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, add, &p, a.descriptor, a.data.u8, &q, b.descriptor, b.data.u8, &zero, c.descriptor, c.data.u8));
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, add);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(c);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_add_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const float p = cmd.info.blas.a[0];
	const float q = cmd.info.blas.a[1];
	if (inputs[0] == 0)
	{
		if (outputs[0])
		{
			const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
			CUDNN_ENFORCE(hipdnnSetTensor(cudnn, a.descriptor, a.data.u8, &p));
			ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
		}
		if (output_size > 1 && outputs[1])
		{
			const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[1]);
			CUDNN_ENFORCE(hipdnnSetTensor(cudnn, b.descriptor, b.data.u8, &q));
			ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
		}
		return CCV_NNC_EXEC_SUCCESS;
	}
	int gdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_t g = ccv_nnc_get_tensor_view(inputs[0]);
	ccv_nnc_tensor_view_get_dim(&g, gdim);
	static const float zero = 0;
	const ccv_nnc_cudnn_tensor_view_descriptor_t gcu = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &g);
	ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_cudnn_tensor_view_descriptor_t acu;
	if (a)
	{
		ccv_nnc_tensor_view_t atv = ccv_nnc_get_tensor_view(outputs[0]);
		ccv_nnc_tensor_view_t* tvs[] = {
			&atv, &g
		};
		ccv_nnc_tensor_view_alignment(tvs, 2);
		acu = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &atv);
	}
	const int reduce_a_dim = a ? !ccv_nnc_tensor_view_check_dim(a, gdim) : 0;
	ccv_nnc_tensor_view_t* const b = output_size > 1 ? (ccv_nnc_tensor_view_t*)outputs[1] : 0;
	ccv_nnc_cudnn_tensor_view_descriptor_t bcu;
	if (b)
	{
		ccv_nnc_tensor_view_t btv = ccv_nnc_get_tensor_view(outputs[1]);
		ccv_nnc_tensor_view_t* tvs[] = {
			&btv, &g
		};
		ccv_nnc_tensor_view_alignment(tvs, 2);
		bcu = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &btv);
	}
	const int reduce_b_dim = b ? !ccv_nnc_tensor_view_check_dim(b, gdim) : 0;
	hipdnnReduceTensorDescriptor_t reduce_sum;
	if ((a && reduce_a_dim) || (b && reduce_b_dim))
	{
		reduce_sum = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
		hipdnnSetReduceTensorDescriptor(reduce_sum, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	}
	size_t workspace_size = 0;
	void* workspace = 0;
	if (a && reduce_a_dim)
	{
		size_t a_workspace_size = 0;
		CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_sum, gcu.descriptor, acu.descriptor, &a_workspace_size));
		if (a_workspace_size > workspace_size)
			workspace_size = a_workspace_size;
	}
	if (b && reduce_b_dim)
	{
		size_t b_workspace_size = 0;
		CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_sum, gcu.descriptor, bcu.descriptor, &b_workspace_size));
		if (b_workspace_size > workspace_size)
			workspace_size = b_workspace_size;
	}
	if (workspace_size)
		workspace = ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
	if (a)
	{
		if (reduce_a_dim)
		{
			CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_sum, 0, 0, workspace, workspace_size, &p, gcu.descriptor, gcu.data.u8, &zero, acu.descriptor, acu.data.u8));
		} else {
			CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &p, gcu.descriptor, gcu.data.u8,  &zero, acu.descriptor, acu.data.u8));
		}
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(acu);
	}
	if (b)
	{
		if (reduce_b_dim)
		{
			CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_sum, 0, 0, workspace, workspace_size, &q, gcu.descriptor, gcu.data.u8, &zero, bcu.descriptor, bcu.data.u8));
		} else {
			CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &q, gcu.descriptor, gcu.data.u8,  &zero, bcu.descriptor, bcu.data.u8));
		}
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(bcu);
	}
	if ((a && reduce_a_dim) || (b && reduce_b_dim))
		ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce_sum);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(gcu);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_ADD_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_add_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_ADD_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_add_back;
#endif
}

