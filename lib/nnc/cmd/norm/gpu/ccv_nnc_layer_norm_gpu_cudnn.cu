#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

template<typename NUM>
__global__ void _ccv_nnc_inv_std_kernel(const int count, const float epsilon, const NUM* const a, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM)(1. / ((float)a[i] + epsilon));
	}
}

static int _ccv_nnc_layer_norm_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	static const float one = 1, zero = 0, neg_one = -1;
	assert(output_size == 3);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	assert(!CCV_IS_TENSOR_VIEW(inputs[1]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t scale = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[1]);
	assert(!CCV_IS_TENSOR_VIEW(inputs[2]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t bias = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[2]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	assert(!CCV_IS_TENSOR_VIEW(outputs[1]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_mean = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[1]);
	assert(!CCV_IS_TENSOR_VIEW(outputs[2]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_inv_std = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[2]);
	int adim[CCV_NNC_MAX_DIM + 2];
	int rdim[CCV_NNC_MAX_DIM + 2];
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[0], adim);
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[1], rdim);
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)inputs[2], rdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[1], rdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[2], rdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[0], adim));
	int x;
	int n = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n *= adim[x];
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n /= rdim[x];
	int rcount = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		rcount *= rdim[x];
	const float inv_n = 1. / n;
	hipdnnReduceTensorDescriptor_t reduce = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
	size_t saved_mean_workspace_size = 0;
	size_t saved_inv_std_workspace_size = 0;
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, a.descriptor, saved_mean.descriptor, &saved_mean_workspace_size));
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_NORM2, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, b.descriptor, saved_inv_std.descriptor, &saved_inv_std_workspace_size));
	const size_t workspace_size = ccv_max(saved_mean_workspace_size, saved_inv_std_workspace_size);
	void* workspace = 0;
	if (workspace_size)
		workspace = ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &inv_n, a.descriptor, a.data.u8, &zero, saved_mean.descriptor, saved_mean.data.u8));
	hipdnnOpTensorDescriptor_t op = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, a.descriptor, a.data.u8, &neg_one, saved_mean.descriptor, saved_mean.data.u8, &zero, b.descriptor, b.data.u8));
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_NORM2, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	const float inv_n_sqrt = sqrt(inv_n);
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &inv_n_sqrt, b.descriptor, b.data.u8, &zero, saved_inv_std.descriptor, saved_inv_std.data.u8));
	// The epsilon is used a little bit differently from batch norm, it is outside of the sqrt in this case.
	const float epsilon = cmd.info.lnorm.epsilon;
	assert(outputs[2]->info.datatype == CCV_16F || outputs[2]->info.datatype == CCV_32F);
	if (outputs[2]->info.datatype == CCV_16F)
		_ccv_nnc_inv_std_kernel<<<CUDA_GET_BLOCKS(rcount), CUDA_NUM_THREADS, 0, stream>>>(rcount, epsilon, (__half*)saved_inv_std.data.f16, (__half*)saved_inv_std.data.f16);
	else
		_ccv_nnc_inv_std_kernel<<<CUDA_GET_BLOCKS(rcount), CUDA_NUM_THREADS, 0, stream>>>(rcount, epsilon, saved_inv_std.data.f32, saved_inv_std.data.f32);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, b.descriptor, b.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, scale.descriptor, scale.data.u8, &zero, b.descriptor, b.data.u8));
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, bias.descriptor, bias.data.u8, &zero, b.descriptor, b.data.u8));
	ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce);
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, op);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(scale);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_layer_norm_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 9);
	assert(output_size == 3);
	// hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	// static const float one = 1, zero = 0;
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[5]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t scale = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[6]);
	// ccv_nnc_tensor_t* const saved_mean = inputs[13];
	// ccv_nnc_tensor_t* const saved_inv_std = inputs[14];
	// ccv_nnc_tensor_t* const dscale = outputs[1];
	// ccv_nnc_tensor_t* const dbias = outputs[2];
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(scale);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_LAYER_NORM_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_layer_norm_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_LAYER_NORM_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_layer_norm_back;
#endif
}
